#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(Matrix N, Matrix P)
{
	/********************************************************************
	Determine input and output indexes of each thread
	Load a tile of the input image to shared memory
	Apply the filter on the input image tile
	Write the compute values to the output image at the correct indexes
	********************************************************************/

    //INSERT KERNEL CODE HERE
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * TILE_SIZE + ty;
    int col_o = blockIdx.x * TILE_SIZE + tx;

    int row_i = row_o - 2; // Assumes kernel size is 5
    int col_i = col_o - 2; // Assumes kernel size is 5

    float output = 0.0;
    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];
    if((row_i >= 0) && (row_i < N.height) && (col_i >= 0) && (col_i < N.width)) {
      Ns[ty][tx] = N.elements[row_i*N.width + col_i];
    }
    else {
      Ns[ty][tx] = 0.0;
    }

    __syncthreads();

    if(ty < TILE_SIZE && tx < TILE_SIZE){
        for(int i = 0; i < 5; i++)
          for(int j = 0; j < 5; j++)
            output += M_c[i][j] * Ns[i+ty][j+tx];

      if(row_o < P.height && col_o < P.width)
        P.elements[row_o * P.width + col_o] = output;
    }

      __syncthreads();
}
